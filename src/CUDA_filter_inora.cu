
#include <hip/hip_runtime.h>
#ifdef CUDA
/** 
 * @file filter_rlsf_mp.c
 * Routines for RLSF filtering of a color image
 */
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include "math_constants.h"
#include "device_functions.h"

#include <stdio.h>
#include "image.h"
#include "math.h"
/** 
 * @brief Implements the INORA Filter
 *
 * @param[in] in_img Image pointer { rgb }
 * @param[in] block_size Radius of the Block { positive-odd }
 * @param[in] alpha alpha nearest pixels in comptation window { positive }
 * @param[in] h smooting prameter spatial domain { positive }
 * @param[in] sigma2 smoothing prameter for peer group { positive }
 *
 * @return Pointer to the filtered image or NULL
 *
 * @author Kusnik Damian 
 * @date 25.02.2024
*/
namespace CUDA_INORA
{

__device__
float compute_ROAD(int* in_data, int width, float r, float g, float b, int window_pos, int alpha, float sigma) {
	float w, weights[9], r1, g1, b1;

	int f = 1;
	int a = 0;
	for (int i = -f; i <= f; i++)
		for (int j = -f; j <= f; j++)
		{
			//if (i == 0 && j == 0 && pixel_pos==window_pos)
			//	continue;

			r1 = (in_data[window_pos + i * width + j] & 0XFF0000) >> 16;
			g1 = (in_data[window_pos + i * width + j] & 0XFF00) >> 8;
			b1 = (in_data[window_pos + i * width + j] & 0XFF);
			weights[a] = (r - r1) * (r - r1) + (g - g1) * (g - g1) + (b - b1) * (b - b1);
			a++;
		}

	w = 0;

	//this is faster than sorting
	for (int i = 0; (i <= alpha) && (i < a); i++)
	{
		float min = weights[0];
		int tmp = 0;
		for (int j = 1; j < 9; j++)
		{
			if (weights[j] < min)
			{
				min = weights[j];
				tmp = j;
			}
		}
		w += min;
		weights[tmp] = +INFINITY;
	}

	w /= (float)(alpha);
	w = __expf(-(w / sigma));
	return w;
}


__device__
float compute_weight_inora_with_partial(int* in_data, float* partial_road, int width, float r, float g, float b, int pos, int alpha, float sigma, float sigma2, float* central_pix, int iter) {
	float w, weights = 0, r1, g1, b1;

	int f = 1;
	int pos2;
	int a = 0;
	int max_f = 9;
	float sum = 0;

	for (int i = -f; i <= f; i++)
		for (int j = -f; j <= f; j++)
		{
			pos2 = pos + i * width + j;
			r1 = (in_data[pos2] & 0XFF0000) >> 16;
			g1 = (in_data[pos2] & 0XFF00) >> 8;
			b1 = (in_data[pos2] & 0XFF);
			sum += ((r - r1) * (r - r1) + (g - g1) * (g - g1) + (b - b1) * (b - b1)) * partial_road[pos2];
			weights += partial_road[pos2];
			a++;
		}

	if (iter)
	{
		r1 = central_pix[0];
		g1 = central_pix[1];
		b1 = central_pix[2];
		w = compute_ROAD(in_data, width, r1, g1, b1, pos, alpha, sigma2);

		sum  += ((r - r1) * (r - r1) + (g - g1) * (g - g1) + (b - b1) * (b - b1)) * w;
		weights += w;
		a++;
	}

	sum /= weights;
	return  __expf(-(sum / sigma));

}


__global__
void precalculate_pixels_INORA(int* in_data, float* pixels_ROAD, const int width, const int height, const int alpha, const float sigma)
{
	int ic, ir;
	int f = 1;

	ic = blockIdx.y * blockDim.y + threadIdx.y;
	ir = blockIdx.x * blockDim.x + threadIdx.x;
	if (ic >= width - f || ir >= height - f || ic < f || ir < f)
		return;
	int pos = ir * width + ic;
	float r = (in_data[pos] & 0XFF0000) >> 16;
	float g = (in_data[pos] & 0XFF00) >> 8;
	float b = (in_data[pos] & 0XFF);

	float w = compute_ROAD(in_data, width, r, g, b, pos, alpha, sigma);
	pixels_ROAD[pos] = w;
	return;
}

__global__
void denoise_pixel_inora(int* in_data, float* pixels_ROAD, int* out_data, const int width, const int height, const int radius, const int alpha, const float sigma, const float sigma2, const int iter)

{
	int f = 1;
	float wsum = 0.0, w, mx, my, r, g, b, last_ir, last_ic, ic, ir, last_r, last_g, last_b;
	int iter_count = 0;

	ic = blockIdx.y * blockDim.y + threadIdx.y;
	ir = blockIdx.x * blockDim.x + threadIdx.x;
	if (ic >= width - f || ir >= height - f || ic < f || ir < f)
		return;

	int pos = ir * width + ic;
	int out_pos = pos;

	//if we are in the image borders
	//if (ir >= height - r || ic >= width - r) return;
	//if (ir < r || ic < r) return;
	r = (in_data[pos] & 0XFF0000) >> 16;
	g = (in_data[pos] & 0XFF00) >> 8;
	b = (in_data[pos] & 0XFF);


	float diff = 0;

	float central_pix[3];

	// go through all pixels in block
	do {

		int istart = max((int)round(ir) - radius - 1, 1);
		int iend = min((int)round(ir) + radius + 1, height - 2);
		int jstart = max((int)round(ic) - radius - 1, 1);
		int jend = min((int)round(ic) + radius + 1, width - 2);

		wsum = w = 0.0;
		last_ir = ir;
		last_ic = ic;
		last_r = r;
		last_g = g;
		last_b = b;

		central_pix[0] = r;
		central_pix[1] = g;
		central_pix[2] = b;

		r = 0;
		g = 0;
		b = 0;

		wsum = 0;
		mx = 0, my = 0;
		pos = (int)round(ir) * width + (int)round(ic);
		for (int i = istart; i <= iend; i++) { // i = y
			for (int j = jstart; j <= jend; j++) { // j = x
				int q = i * width + j;
				w = compute_weight_inora_with_partial(in_data, pixels_ROAD, width,
					(in_data[q] & 0XFF0000) >> 16,
					(in_data[q] & 0XFF00) >> 8,
					(in_data[q] & 0XFF),
					pos, alpha, sigma, sigma2, central_pix, iter_count);
				w *= pixels_ROAD[q];
				r += ((in_data[q] & 0XFF0000) >> 16)* w;
				g += ((in_data[q] & 0XFF00) >> 8)* w;
				b += (in_data[q] & 0XFF) * w;
				wsum += w;
				//zastanowic sie co robimy z x i y
				mx += i * w;
				my += j * w;

			}
		}

		diff = 0;
		r = r / wsum;
		g = g / wsum;
		b = b / wsum;

		ir = mx / wsum;
		ic = my / wsum;

		if (ir < 0)
			ir = 0;
		if (ic < -0)
			ic = 0;
		diff = (last_r - r) * (last_r - r) + (last_g - g) * (last_g - g) + (last_b - b) * (last_b - b)
			+ (last_ir - ir) * (last_ir - ir) + (last_ic - ic) * (last_ic - ic);
		iter_count++;
	} while (iter_count < iter && diff >0);

	out_data[out_pos] = ((int)(r) << 16) |
		((int)(g) << 8) |
		((int)(b));
	return;
}
}

Image *
CUDA_filter_inora ( const Image * in_img, const int r, int alpha, const float sigma, const float sigma2, const int iter)
{
 using namespace CUDA_INORA;
 SET_FUNC_NAME ( "filter_inora" );

 byte*** in_data;
 byte*** out_data;
 int num_rows, num_cols;
 Image* out_img;
 if ( !is_rgb_img ( in_img ) )
  {
   ERROR_RET ( "Not a color image !", NULL );
  }

 if ( !IS_POS ( r ) )
  {
   ERROR ( "Window size ( %d ) must be positive !", r );
   return NULL;
  }

 if ( !IS_POS ( alpha ) )
  {
   ERROR ( "Alpha value ( %d ) must be positive !", alpha );
   return NULL;
  }

 if ( !IS_POS ( sigma ) )
  {
   ERROR ( "Sigma value ( %d ) must be positive !", sigma );
   return NULL;
  }

 if (!IS_POS(sigma2))
 {
	 ERROR("Sigma2 value ( %d ) must be positive !", sigma2);
	 return NULL;
 }


 num_rows = get_num_rows(in_img);
 num_cols = get_num_cols(in_img);

 in_data = (byte***)get_img_data_nd(in_img);
 out_img = alloc_img(PIX_RGB, num_rows, num_cols);
 out_data = (byte***)get_img_data_nd(out_img);

 //	cudaProfilerStart();


 //size_t size_b = size_t(num_rows * num_cols) * sizeof(byte);
 size_t size_i = size_t(num_rows * num_cols) * sizeof(int);
 size_t size_f = size_t(num_rows * num_cols) * sizeof(float);

 int* int_in_data = (int*)malloc(size_i);
 for (int i = 0; i < num_rows; i++) {
	 for (int j = 0; j < num_cols; j++)
	 {
		 int_in_data[i * num_cols + j] = (((int)in_data[i][j][0]) << 16) | ((int)in_data[i][j][1] << 8) | ((int)in_data[i][j][2]);
	 }
 }

 int* d_in_data;
 cudaMalloc((void**)&d_in_data, size_i);
 cudaMemcpy(d_in_data, int_in_data, size_i, cudaMemcpyHostToDevice);

 int* d_int_out_data;
 cudaMalloc((void**)&d_int_out_data, size_i);

 float* d_pixels_ROAD;
 cudaMalloc((void**)&d_pixels_ROAD, size_f);
 cudaMemset(d_pixels_ROAD, 0, size_f);

 dim3 blockDim(1, 128, 1);
 dim3 gridDim((unsigned int)ceil((float)num_rows / (float)blockDim.x),
	 (unsigned int)ceil((float)num_cols / (float)blockDim.y),
	 1);

 precalculate_pixels_INORA << < gridDim, blockDim >> > (d_in_data, d_pixels_ROAD, num_cols, num_rows, alpha, 2 * sigma2 * sigma2);
 cudaDeviceSynchronize();
 denoise_pixel_inora << < gridDim, blockDim >> > (d_in_data, d_pixels_ROAD, d_int_out_data, num_cols, num_rows, r, alpha, 2 * sigma * sigma, 2 * sigma2 * sigma2, iter);
 cudaDeviceSynchronize();

 int* int_out_data = (int*)malloc(size_i);
 cudaMemcpy(int_out_data, d_int_out_data, size_i, cudaMemcpyDeviceToHost);


 for (int i = 0; i < num_rows; i++)
	 for (int j = 0; j < num_cols; j++)
	 {
		 out_data[i][j][0] = (int_out_data[i * num_cols + j] >> 16) & 0xFF;
		 out_data[i][j][1] = (int_out_data[i * num_cols + j] >> 8) & 0xFF;
		 out_data[i][j][2] = (int_out_data[i * num_cols + j]) & 0xFF;

	 }

 // Free device memory

 cudaFree(d_in_data);
 cudaFree(d_int_out_data);
 cudaFree(d_pixels_ROAD);
 cudaDeviceSynchronize();

 free(int_in_data);
 free(int_out_data);

 return out_img;
}
#endif
